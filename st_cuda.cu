#include "hip/hip_runtime.h"
/*
	Programa: st_cuda.c  (Versão 2)
	Descrição: Implementa o Algoritmo para árvore geradora.
	Programadora: Jucele Vasconcellos
	Data: 28/07/2017

	Compilacao:	nvcc -arch sm_30 -o st_cuda.exe st_cuda.cu
	Execucao:	./st_cuda.exe in/grafo/grafo1000a cuda.out
	
	Entrada de dados: Este programa lê os dados de um grafo no formato
	8
	16
	4 5 0.35
	4 7 0.37
	5 7 0.28
	0 7 0.16
	1 5 0.32
	0 4 0.38
	2 3 0.17
	1 7 0.19
	0 2 0.26
	1 2 0.36
	1 3 0.29
	2 7 0.34
	6 2 0.40
	3 6 0.52
	6 0 0.58
	6 4 0.93

	sendo a primeira linha o número de vértices, a segunda linha o número de arestas
	e as linhas subsequentes as arestas v1 v2 custo
		
	Saída de Dados: Este programa produz um arquivo de saída as arestas que compõem a árvore geradora 
*/

#include <stdio.h> // printf
#include<stdbool.h> // true, false
#include <stdlib.h> //malloc
#include <time.h> //clock
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

// Grafo Original
typedef struct { 
	unsigned int v, u;
	unsigned int grau_st;
} aresta;


typedef struct { 
	int n, m;
	aresta *arestas;
} grafo;

typedef struct {
	int v1, v2;
} aresta_E;


// Funções e Procedimentos

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line );
        exit(EXIT_FAILURE);
    }
}

#define CHECK_ERROR(err) (HandleError(err, __FILE__, __LINE__))

grafo LeGrafo(char *);
__global__ void EncontraMenorAresta1(aresta*, int*, int, int);
__global__ void EncontraMenorAresta2(aresta*, int*, int, int);
__global__ void MarcarArestas_Strut(aresta*, int*, int);
__global__ void Calcula_num_zerodiff(aresta*, int, int*, unsigned int*, unsigned int*);
__global__ void Inicializa_arestasE_C(aresta*, int*, int, int*, aresta_E*, unsigned int*);
__global__ void AtualizaC_1(aresta_E*, int*, int*, int); 
__global__ void DefineNovosVU(int*, int, char *, unsigned int *); 
__global__ void AtualizaC_3(int*, int, char *); 
__global__ void MarcarArestas(aresta *, int*, int, int);



// Função Principal
int main (int argc, char** argv){
	grafo G;
	double tempoTotal, tempo1, tempo2;
	double tempo1p, tempo2p;
	int *SolutionEdgeSet;
	int SolutionSize, i, it;
	int num_zerodiff;
	FILE *Arq;
		
	int dimBloco, dimGrid;
	aresta *d_arestas;
	int *d_SolutionEdgeSet;
	unsigned int *d_SolutionSize, *d_num_zerodiff;
	int *d_menorAresta;
	int n;
	
	
	// Passo 1: Verificação de parâmetros
	// Passo 2: Leitura dos dados do grafo 
	// Passo 3: Criação do grafo bipartido correspondente às arestas recebidas
	// Passo 4: Encontra a solução
		// Passo 4.1: Escolher arestas que comporão a strut
		// Passo 4.2: Calcular o num_zerodiff e computar novas componenetes conexas
		// Passo 4.3: Compactar o grafo
	
	
	// ==============================================================================
	// Passo 1: Verificação de parâmetros
	// ==============================================================================
	
	//Verificando os parametros
	if(argc < 3 ){
	   printf( "\nParametros incorretos\n Uso: ./cms_seq.exe <ArqEntrada> <ArqSaida> <dimBloco> <S/N> onde:\n" );
	   printf( "\t <ArqEntrada> (obrigatorio) - Nome do arquivo com as informações do grafo (número de vértices, número de arestas e arestas.\n" );
		printf( "\t <ArqSaida> (obrigatorio) - Nome do arquivo de saida.\n" );
		printf( "\t <S ou N> - Mostrar ou não as arestas da MST.\n" );
		return 0;
	} 	
	//Define a dimensão do bloco fixada em 32 threads
	dimBloco = 32;
	
	// ==============================================================================
	// Passo 2: Leitura dos dados do Grafo G
	// ==============================================================================
	tempo1p = (double) clock( ) / CLOCKS_PER_SEC;
	G = LeGrafo(argv[1]);
	n = G.n;
// 	printf("****************************\n");
// 	printf("****** GRAFO ORIGINAL ******\n");
// 	MostraGrafoOriginal(GO);
	
// 	printf("Grafo de entrada lido\n");
	
	//Alocação de variável para armazenar solução
	SolutionEdgeSet = (int *) malloc((G.n-1)*sizeof(int)); 
	//Aloca memória no device para as arestas da d_SolutionEdgeSet
	CHECK_ERROR(hipMalloc((void **) &d_SolutionEdgeSet, G.n * sizeof(int)));
	SolutionSize = 0;
	//Aloca memória no device para a variável d_SolutionSize
	CHECK_ERROR(hipMalloc((void **) &d_SolutionSize, sizeof(unsigned int)));
	//Inicializa d_SolutionSize com 0
 	CHECK_ERROR(hipMemset(d_SolutionSize, 0, sizeof(unsigned int)));
	//Aloca memória no device para a variável d_num_zerodiff
	CHECK_ERROR(hipMalloc((void **) &d_num_zerodiff, sizeof(unsigned int)));

	tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
// 	printf("Tempo Passo 2: %lf\n", tempo2p - tempo1p);
	
	// ==============================================================================
	// Passo 3: Tranfere informações para device e inicializa variáveis
	// ==============================================================================
	//Iniciando contagem do tempo
	tempo1 = (double) clock( ) / CLOCKS_PER_SEC;
	tempo1p = (double) clock( ) / CLOCKS_PER_SEC;
	
	//Aloca memória no device para as arestas do grafo
	CHECK_ERROR(hipMalloc((void **) &d_arestas, G.m * sizeof(aresta)));
	//Copia as arestas do grafo do host para o device
	CHECK_ERROR(hipMemcpy(d_arestas, G.arestas, G.m * sizeof(aresta), hipMemcpyHostToDevice));
	
	//Aloca memória no device para o vetor d_menorAresta
	CHECK_ERROR(hipMalloc((void **) &d_menorAresta, G.n * sizeof(int)));
	
	
	tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
// 	printf("Tempo Passo 3: %lf\n", tempo2p - tempo1p);
	
	// ==============================================================================
	// Passo 4: Encontra solução
	// ==============================================================================
	
	it = 0;
	num_zerodiff = 0;
	while(num_zerodiff != 1)
	{
		// ==============================================================================
		// Passo 4.1: Escolher arestas que comporão a strut
		// ==============================================================================
		tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

		//Inicializa d_menorAresta com -1
		CHECK_ERROR(hipMemset(d_menorAresta, -1, G.n * sizeof(int)));
		
		//Chama kernel para encontrar menorAresta de cada v
		dimGrid = ((G.m-1)/dimBloco)+1;
		EncontraMenorAresta1<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, G.m, G.n);
// 		if(it >= 0)
// 		{
// 			printf("Após EncontraMenorAresta1\n");
// 			int *h_menorAresta;
// 			h_menorAresta = (int *) malloc(G.n * sizeof(int)); 
// 			CHECK_ERROR(hipMemcpy(h_menorAresta, d_menorAresta, G.n * sizeof(int), hipMemcpyDeviceToHost));
// 			for(i = 0; i < n; i++)
// 				printf("MenorAresta[%d] = %d\n", i, h_menorAresta[i]);
// 			free(h_menorAresta);
// 		}
		dimGrid = ((G.m-1)/dimBloco)+1;
		EncontraMenorAresta2<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, G.m, G.n);
// 		if(it >= 0)
// 		{
// 			printf("Após EncontraMenorAresta2\n");
// 			int *h_menorAresta;
// 			h_menorAresta = (int *) malloc(G.n * sizeof(int)); 
// 			CHECK_ERROR(hipMemcpy(h_menorAresta, d_menorAresta, G.n * sizeof(int), hipMemcpyDeviceToHost));
// 			for(i = 0; i < n; i++)
// 				printf("MenorAresta[%d] = %d\n", i, h_menorAresta[i]);
// 			free(h_menorAresta);
// 		}
		
		
		//Chama kernel para marcar arestas da Strut
		dimGrid = ((n-1)/dimBloco)+1;
		MarcarArestas_Strut<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, n);
		
		tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
// 		printf("Tempo Passo 4.1: %lf\n", tempo2p - tempo1p);
		
		// ==============================================================================
		// Passo 4.2: Calcular o num_zerodiff
		// ==============================================================================
		tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

		//Inicializa d_num_zerodiff com 0
		CHECK_ERROR(hipMemset(d_num_zerodiff, 0, sizeof(unsigned int)));
	
		//Chama kernel para calcular num_zerodiff e preencher SolutionEdgeSet
		dimGrid = ((G.m-1)/dimBloco)+1;
		Calcula_num_zerodiff<<<dimGrid, dimBloco>>>(d_arestas, G.m, d_SolutionEdgeSet, d_SolutionSize, d_num_zerodiff);
		
		CHECK_ERROR(hipMemcpy(&num_zerodiff, d_num_zerodiff, sizeof(unsigned int), hipMemcpyDeviceToHost));
 
		tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
//  		printf("Tempo Passo 4.2: %lf     it = %d     num_zerodiff = %d     SolutionSize = %d\n", tempo2p - tempo1p, it, num_zerodiff, SolutionSize);
		
		// ==============================================================================
		// Passo 4.3: Compactar o grafo
		// ==============================================================================
		if(num_zerodiff != 1)
		{
			// ==============================================================================
			// Passo 4.3.1: Computar componenetes conexas
			// ==============================================================================
			tempo1p = (double) clock( ) / CLOCKS_PER_SEC;

			//Declara variável d_aux
			unsigned int *d_aux;
			//Aloca memória no device para a variável d_aux
			CHECK_ERROR(hipMalloc((void **) &d_aux, sizeof(unsigned int)));
			//Inicializa d_aux com 0
			CHECK_ERROR(hipMemset(d_aux, 0, sizeof(unsigned int)));

			//Declara variável d_arestasE
			aresta_E *d_arestasE;
			//Aloca memória no device para a variável d_arestasE
			CHECK_ERROR(hipMalloc((void **) &d_arestasE, n * sizeof(aresta_E)));

			//Declara variável d_CD
			int *d_C;
			//Aloca memória no device para a variável d_CD
			CHECK_ERROR(hipMalloc((void **) &d_C, n * sizeof(int)));
			
			//Chama kernel para inicializar d_arestasE e d_C
			dimGrid = ((n-1)/dimBloco)+1;
			Inicializa_arestasE_C<<<dimGrid, dimBloco>>>(d_arestas, d_menorAresta, n, d_C, d_arestasE, d_aux);
			
			int h_fim, *d_fim;
			CHECK_ERROR(hipMalloc((void**)&d_fim, sizeof(int)));
			dimGrid = ((n-1-num_zerodiff)/dimBloco)+1;
			do
			{
				h_fim = 0;
				CHECK_ERROR(hipMemcpy(d_fim, &h_fim, sizeof(int), hipMemcpyHostToDevice));
				AtualizaC_1<<<dimGrid, dimBloco>>>(d_arestasE, d_C, d_fim, n-num_zerodiff);
				CHECK_ERROR(hipMemcpy(&h_fim, d_fim, sizeof(int), hipMemcpyDeviceToHost));		
			}while (h_fim == 1);
			
			CHECK_ERROR(hipDeviceSynchronize());
			
			//Declara variável d_aux2
			char *d_aux2;
			//Aloca memória no device para a variável d_aux2
			CHECK_ERROR(hipMalloc((void **) &d_aux2, n * sizeof(char)));
			//Inicializa d_aux2 com 0
			CHECK_ERROR(hipMemset(d_aux2, 0, n * sizeof(char)));
			//Inicializa d_aux com 0
			CHECK_ERROR(hipMemset(d_aux, 0, sizeof(unsigned int)));
			//Chama kernel para atualizar d_C
			dimGrid = ((n-1)/dimBloco)+1;
// 			printf("Vou chamar DefineNovosVU para dimGrid = %d  e  dimBloco = %d   com n = %d\n", dimGrid, dimBloco, n);
			DefineNovosVU<<<dimGrid, dimBloco>>>(d_C, n, d_aux2, d_aux);
			
			CHECK_ERROR(hipDeviceSynchronize());
			
			//Chama kernel para atualizar d_C
			dimGrid = ((n-1)/dimBloco)+1;
			AtualizaC_3<<<dimGrid, dimBloco>>>(d_C, n, d_aux2);
			
			//Liberando variáveis
			CHECK_ERROR(hipFree(d_aux));
			CHECK_ERROR(hipFree(d_arestasE));
			CHECK_ERROR(hipFree(d_fim));
			CHECK_ERROR(hipFree(d_aux2));

			tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
// 			printf("Tempo Passo 4.3.1: %lf\n", tempo2p - tempo1p);
			
			// ==============================================================================
			// Passo 4.3.2: Marcar arestas
			// ==============================================================================
			tempo1p = (double) clock( ) / CLOCKS_PER_SEC;
			
			//Marca as arestas para remoção
			dimGrid = ((G.m-1)/dimBloco)+1;
			MarcarArestas<<<dimGrid, dimBloco>>>(d_arestas, d_C, G.m, G.n);
			
			CHECK_ERROR(hipDeviceSynchronize());
			
			CHECK_ERROR(hipFree(d_C));
			
			
// 			aresta *h_arestas;
// 			h_arestas = (aresta *) malloc(G.m * sizeof(aresta)); 
// 			CHECK_ERROR(hipMemcpy(h_arestas, d_arestas, G.m * sizeof(aresta), hipMemcpyDeviceToHost));
// 			for(i = 0; i < G.m; i++)
// 				if(h_arestas[i].v != G.n)
// 					printf("Aresta[%d]     v = %d    u = %d\n", i, h_arestas[i].v, h_arestas[i].u);
// 			free(h_arestas);
			
						
			tempo2p = (double) clock( ) / CLOCKS_PER_SEC;
// 			printf("Tempo Passo 4.3.2: %lf\n", tempo2p - tempo1p);		
		}

		it++;
		n = num_zerodiff;
	} // fim while(num_zerodiff != 1)
	
	
	// Copia a Solução para o host
	CHECK_ERROR(hipMemcpy(&SolutionSize, d_SolutionSize, sizeof(unsigned int), hipMemcpyDeviceToHost));
	CHECK_ERROR(hipMemcpy(SolutionEdgeSet, d_SolutionEdgeSet, (G.n-1) * sizeof(int), hipMemcpyDeviceToHost));
	
	tempo2 = (double) clock( ) / CLOCKS_PER_SEC;
	tempoTotal = tempo2 - tempo1;

	printf("%lf\n", tempoTotal);

	Arq = fopen(argv[2], "a");
	fprintf(Arq, "\n*** Arquivo de entrada: %s\n", argv[1]);
	fprintf(Arq, "Tempo Total: %lf\n", tempoTotal); 
	fprintf(Arq, "Número de iterações: %d\n", it);
	fprintf(Arq, "SolutionSize: %d\n", SolutionSize);

  	if((argc == 4) && (argv[3][0] == 'S' || argv[3][0] == 's'))
	{
  		fprintf(Arq, "*** ST formada pelas %d arestas\n", SolutionSize);
  		for(i = 0; i < SolutionSize; i++)
  			fprintf(Arq, "Aresta %d - %d\n", G.arestas[SolutionEdgeSet[i]].v, G.arestas[SolutionEdgeSet[i]].u);
  	}
  	fclose(Arq);

	
	// Liberando variávais alocadas no device
	CHECK_ERROR(hipFree(d_SolutionEdgeSet));
	CHECK_ERROR(hipFree(d_SolutionSize));
	CHECK_ERROR(hipFree(d_num_zerodiff));
	CHECK_ERROR(hipFree(d_arestas));
	CHECK_ERROR(hipFree(d_menorAresta));
	
	// Liberando variávais alocadas no host
	free(G.arestas);
	
	return 0;

}


// ==============================================================================
// Função LeGrafo:  Lê as informações do Grafo de um arquivo e armazena em uma 
//                  estrutura
// ==============================================================================
grafo LeGrafo(char *Arquivo){
	int i, aux;
	grafo G;
	float custo;
   FILE *Arq;
    
   Arq = fopen(Arquivo, "r");

   i = 0;
	fscanf(Arq,"%d",&i);
	G.n = i;
	
	fscanf(Arq,"%d",&i);
	G.m = i;
	
	G.arestas = (aresta *) malloc(G.m*sizeof(aresta)); 
	
	for(i = 0; i < G.m; i++){
		fscanf(Arq,"%hu",&G.arestas[i].u);
		fscanf(Arq,"%hu",&G.arestas[i].v);
		if(G.arestas[i].v > G.arestas[i].u)
		{
			aux = G.arestas[i].v;
			G.arestas[i].v = G.arestas[i].u;
			G.arestas[i].u = aux;
		}
		fscanf(Arq,"%f",&custo);
		G.arestas[i].grau_st = 0;
	}
	
	fclose(Arq);
   return G;
}


// ==============================================================================
// Função EncontraMenorAresta1:  Para cada vértice v encontra o id da aresta menor
// ==============================================================================
__global__ void EncontraMenorAresta1(aresta* arestas, int* menorAresta, int m, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int x, aux;
	int old;
	
	if( id < m )
	{
		if(arestas[id].v != n)
		{
			x = arestas[id].v;
			if((menorAresta[x] == -1) || (menorAresta[x] > id))
			{
				aux = -1;
				// atomicCAS = atomic Compare And Swap
				// lê o conteúdo endereçado por menorAresta[x] e o armazena em old.  
				// computa (old == aux ? meu_id: old)
				// ou seja se (old == aux) então menorAresta[x] = id
				//         senão menorAresta[x] = old
				// retorna old
				old = atomicCAS(&menorAresta[x], aux, id);		
				if(old != aux)
				{		
					while((old > id) && (old != aux))
					{
						aux = atomicCAS(&menorAresta[x], old, id);
						if(old != aux)
						{
							old = aux;
							aux = -1;
						}
					}// fim while((old > id) && (old != aux))
				} // fim if(old_id != aux)
			} // fim if((menorAresta[x] == -1) || (menorAresta[x] > id))
		} // fim if(arestas[id].v != n)
	} //fim if( id < m )
}

// ==============================================================================
// Função EncontraMenorAresta2:  Para cada vértice v encontra o id da aresta menor
// ==============================================================================
__global__ void EncontraMenorAresta2(aresta* arestas, int* menorAresta, int m, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int x, aux;
	int old;
	
	if( id < m )
	{
		if(arestas[id].v != n)
		{
			x = arestas[id].u;
			if((menorAresta[x] == -1) || (menorAresta[x] > id))
			{
				aux = -1;
				// atomicCAS = atomic Compare And Swap
				// lê o conteúdo endereçado por menorAresta[x] e o armazena em old.  
				// computa (old == aux ? meu_id: old)
				// ou seja se (old == aux) então menorAresta[x] = id
				//         senão menorAresta[x] = old
				// retorna old
				old = atomicCAS(&menorAresta[x], aux, id);		
				if(old != aux)
				{
					while((old > id) && (old != aux))
					{
						aux = atomicCAS(&menorAresta[x], old, id);
						if(old != aux)
						{
							old = aux;
							aux = -1;
						}
					}// fim while((old > id) && (old != aux))
				} // fim if(old_id != aux)
			} // if((menorAresta[x] == -1) || (menorAresta[x] > id))
		} // fim if(arestas[id].v != n)
	} //fim if( id < m )
}


// ==============================================================================
// Função MarcarArestas_Strut:  Para cada vértice v marca aresta pertencente a strut
// ==============================================================================
__global__ void MarcarArestas_Strut(aresta* d_arestas, int* d_menorAresta, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if( id < n )
	{
		atomicInc(&d_arestas[d_menorAresta[id]].grau_st, UINT_MAX);
	}
}


// ==============================================================================
// Função Calcula_num_zerodiff:  Calcula o número de vértices zero diferença e
//                               preenche o vetor d_SolutionEdgeSet
// ==============================================================================
__global__ void Calcula_num_zerodiff(aresta* arestas, int m, int* SolutionEdgeSet, unsigned int* SolutionSize, unsigned int* num_zerodiff)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int pos;
	
	if( id < m )
	{
		if(arestas[id].grau_st > 0)
		{
			pos = atomicInc(&SolutionSize[0], UINT_MAX);
			SolutionEdgeSet[pos] = id;
			if(arestas[id].grau_st == 2)
			{
				pos = atomicInc(&num_zerodiff[0], UINT_MAX);
// 				printf("num_zerodiff = %d\n", num_zerodiff[0]);
			}
		}
	}
}




// ==============================================================================
// Função Inicializa_arestasE_CD:  Inicializa as variáveis CD, arestasE e arestasE_size
// ==============================================================================
__global__ void Inicializa_arestasE_C(aresta *arestas, int *menorAresta, int n, int *C, aresta_E *arestasE, unsigned int *arestasE_size)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int pos;
	
	if( id < n )
	{
		if((arestas[menorAresta[id]].grau_st == 1) || (((arestas[menorAresta[id]].grau_st == 2)) && (id == arestas[menorAresta[id]].v)))
		{
			pos = atomicInc(&arestasE_size[0], UINT_MAX);
			arestasE[pos].v1 = arestas[menorAresta[id]].v;
			arestasE[pos].v2 = arestas[menorAresta[id]].u;
// 			printf("Thread %d    arestasE[%d]   v1 = %d   v2 = %d\n", id, pos, arestasE[pos].v1, arestasE[pos].v2);
		}
		C[id] = id;
	}
}


// ==============================================================================
// Função AtualizaC_1: Atualiza vetor C para definição das componentes conexas
// ==============================================================================
__global__ void AtualizaC_1(aresta_E *arestasE, int* C, int *m, int n) 
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int c1, c2, v1, v2;
	
	if(id < n)
	{
		v1 = arestasE[id].v1;
		v2 = arestasE[id].v2;
		c1 = C[v1];
		c2 = C[v2];
		if(c1 < c2)
		{
			atomicMin(&C[v2], c1);
			m[0] = 1;
		}
		else if(c2 < c1)
		{
			atomicMin(&C[v1], c2);
			m[0] = 1;
		}
	}
}


// ==============================================================================
// Função DefineNovosVU: Atualiza vetor C
// ==============================================================================
__global__ void DefineNovosVU(int* C, int n, char *marcador, unsigned int *num_comp) 
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int pos;
	
	if(id < n)
	{
		if(C[id] == id)
		{
			pos = atomicInc(&num_comp[0], UINT_MAX);
			C[id] = pos;
			marcador[id] = 1;
			//printf("C2[%d] = %d\n", id, C[id]);
		}
	}	
}


// ==============================================================================
// Função AtualizaC_3: Atualiza vetor C
// ==============================================================================
__global__ void AtualizaC_3(int* C, int n, char *marcador) 
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	
	if(id < n)
	{
		if(marcador[id] == 0)
		{
			C[id] = C[C[id]];
 			//printf("C3[%d] = %d\n", id, C[id]);
		}
	}	
}


// ==============================================================================
// Função MarcarArestas:  Marca as arestas do grafo a serem removidas
// ==============================================================================
__global__ void MarcarArestas(aresta *arestas, int* C, int m, int n)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	int x, y;
	
	if( id < m )
	{
		if(arestas[id].v != n)
		{
			x = C[arestas[id].v];
			y = C[arestas[id].u];
		
			if( x !=  y )
			{
				arestas[id].v = x;
				arestas[id].u = y;
			}
			else
			{
				arestas[id].v = n;
				arestas[id].u = n;
			}
			arestas[id].grau_st = 0;
		}
	} //fim if( id < m )
}


